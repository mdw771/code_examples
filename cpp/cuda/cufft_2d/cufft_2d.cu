/* 2D FFT using CuFFT. */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <cstdlib>

using namespace std;
typedef float2 Complex;

void print_array_complex(Complex* arr, int ny, int nx)
{
    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            cout << arr[iy * nx + ix].x << " + " << arr[iy * nx + ix].y << "j ";
        }
        cout << endl;
    }
}

int main()
{
    int ny = 5;
    int nx = 5;
    int size = ny * nx;
    Complex *img, *res;
    hipfftComplex *img_dev;

    img = new Complex[size];
    res = new Complex[size];
    for (int i = 0; i < size; i++)
    {
        //img[i].x = (float)rand() / (float)RAND_MAX;
        //img[i].y = (float)rand() / (float)RAND_MAX;
        img[i].x = 5.0;
        img[i].y = 0.0;
    }
    
    print_array_complex(img, ny, nx);
    cout << "=======================" << endl;
    
    hipMalloc(&img_dev, sizeof(hipfftComplex) * size);
    hipMemcpy(img_dev, img, sizeof(hipfftComplex) * size, hipMemcpyHostToDevice);
    
    hipfftHandle plan;
    hipfftPlan2d(&plan, ny, nx, HIPFFT_C2C);
    
    hipfftExecC2C(plan, (hipfftComplex *)img_dev, (hipfftComplex *)img_dev, HIPFFT_FORWARD);
    
    hipMemcpy(res, img_dev, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);
    
    print_array_complex(res, ny, nx);
    
    hipfftDestroy(plan);
    hipFree(img_dev);
}